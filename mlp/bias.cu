#include "hip/hip_runtime.h"

#include "bias.h"

__global__ void bias_par(float* input_vector, size_t N, float* bias_vector, float* output_vector);

int bias(float* input_vector, size_t N, float* bias_vector, float* output_vector) {

    // device variables
    float* d_input_vector; 
    float* d_bias_vector;
    float* d_output_vector;

    int num_blks = N / NTHREADS;

    // device memory reserve
    hipMalloc(&d_input_vector, N*sizeof(float));
    hipMalloc(&d_bias_vector, N*sizeof(float));
    hipMalloc(&d_output_vector, N*sizeof(float));

    // device memory initialitation
    hipMemcpy(d_input_vector, input_vector, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bias_vector, bias_vector, N*sizeof(float), hipMemcpyHostToDevice);

    // kernel call
    bias_par<<<num_blks, NTHREADS>>>(d_input_vector, N, d_bias_vector, d_output_vector);
    
    // results recovery
    hipMemcpy(output_vector, d_output_vector, N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input_vector); 
    hipFree(d_bias_vector);
    hipFree(d_output_vector);

    return 0;

}

// GPU Kernel for the bias addition
__global__ void bias_par(float* input_vector, size_t N, float* bias_vector, float* output_vector) {

    int th_id = threadIdx.x + blockDim.x * blockIdx.x; 
    output_vector[th_id] = input_vector[th_id] + bias_vector[th_id];

} 